
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <cstring>
#include <string>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <vector>
#include <set>
#include <iterator>
#include <algorithm>
#include <time.h>

using namespace std;

// Training image file name
const string training_image_fn = "train-images.idx3-ubyte";

// Training label file name
const string training_label_fn = "train-labels.idx1-ubyte";

int classes = 10;

__global__
void saxpy(float n, float a, float *x, float *w, float *sum)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	//printf("%d", index);
	int classes = 10;
	for (int i = index; i < n; i += stride)
		for(int k = 0; k < classes; k++) {
			sum[i + k * (int)n] = w[i + k * (int)n]*x[i] + a;
			//sum[i + k * (int)n] = intermediateW[i + k * (int)n];
		}
}

__global__
void sum_cuda(float n, float *sum, float *total, int run)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	//printf("Index --- %d", index);
	//int classes = 1;
	for (int idx = index; idx < n; idx += stride) {
		//for(int k = 0; k < classes; k++) {
		register int i = atomicAdd(&total[0], sum[idx + run * (int)n]);
		sum[i + run * (int)n] = idx;
		//}
	}
	//for (int idx = index; idx < classes; idx += stride) {
	//printf("i = %d %f\n", i, sum[i]);
	//		for(int k = 0; k < n; k++) {
	//			//printf("i = %d %f\n",i, sum[i]);
	//			//sum[i] += w[i + k * (int)n];
	//			sum[i] += w[i*(int)n + k];
	//			//printf("%f\n",sum[i]);
	//		}
	//	register int i = atomicAdd(total, sum[idx]);
	//	sum[i] = idx;
	//printf("cuda --- %f\n",sum[i]);
	//}
}

__global__
void updateWeights(float n, float *err, float *w, float *x)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	int classes = 10;
	//float a;
	for (int i = index; i < n; i += stride)
		for(int k = 0; k < classes; k++) {
			//printf(" %f  ", w[i + k * (int)n] );
			//a = w[i + k * (int)n];
			w[i + k * (int)n] += (-0.001 * err[k]) * x[i];
			//printf(" %f  after %f changes required %f\n", a, w[i + k * (int)n], err[k] );
		}
	//printf(" after changes required %f\n", err[0] );
	//theta[m2][n] += (alpha * (labelTrain[j][m2] - prob[m2]) * dataTrain[j][n]);
}

void softMax(float *sum)
{
	float total = 0.0f;
	for (int i = 0; i < classes; i += 1)
		total += exp(sum[i]);
	for (int i = 0; i < classes; i += 1)
		sum[i] = exp(sum[i]) / total;
}

// Software: Training Artificial Neural Network for MNIST database
// Author: Hy Truong Son
// Major: BSc. Computer Science
// Class: 2013 - 2016
// Institution: Eotvos Lorand University
// Email: sonpascal93@gmail.com
// Website: http://people.inf.elte.hu/hytruongson/
// Copyright 2015 (c). All rights reserved.

// File stream to read data (image, label) and write down a report
ifstream image;
ifstream label;
ofstream report;

// Number of training samples
const int nTraining = 1;

// Image size in MNIST database
const int width = 28;
const int height = 28;

// Image. In MNIST: 28x28 gray scale images.
int d[width][height];

char inputNum;




void input() {
	// Reading image
	for(int i = 0; i < 1; i++ ) {

		for (int j = 0; j < height; ++j) {
			for (int i = 0; i < width; ++i) {
				image.read(&inputNum, sizeof(char));
				if (inputNum == 0) {
					d[i][j] = 1;
				} else {
					d[i][j] = 0;
				}
			}
		}
		label.read(&inputNum, sizeof(char));
		cout << "Label:" << (int)inputNum << endl;
	}
}

void check(float *sum, int N){
	float total = 0.0f;
	for(int j = 0; j < N; j++)
		total += sum[j];

	cout<<total<< endl;
}

int main(void)
{
	float *x, *d_x, *d_w, *w, *sum, *d_sum;
	//float total = 0, *d_total = 0;
	float *d_index = 0;
	float *h_index = 0;
	float err[10], *d_err;

	int N = width * height;

	cout << "Starting code....... 124"  << endl;

	x = (float *)malloc( N * sizeof(float));
	w = (float *)malloc( N * classes * sizeof(float));
	sum = (float *)malloc( N * classes * sizeof(float));
	h_index = (float *)malloc( classes * sizeof(float));
	//total = (float *)malloc( classes * sizeof(float) );

	for(int i = 0; i < classes; i++)
		h_index[i] = 0;


	/*********** initializing wights *******************/
	for (int i = 0; i < N; i++) {
		for(int j = 0; j < classes; j++)
			w[i + j * N] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
	}

	/*************Transfer Data from host to device *********************/
	hipMalloc(&d_x, N * sizeof(float));
	hipMalloc(&d_w, N * classes * sizeof(float));
	hipMalloc(&d_sum, N * classes * sizeof(float));
	//cudaMalloc(&d_total, classes * sizeof(float));

	hipMalloc(&d_err, classes * sizeof(float));
	hipMalloc( (void**) &d_index, classes * sizeof(float) );
	hipMemcpy(d_w, w, N * classes * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_sum, sum, N * classes * sizeof(float), hipMemcpyHostToDevice);
	//cudaMemcpy(d_total, total, sizeof(float), cudaMemcpyHostToDevice);
	/*************Transfer Data from host to device *********************/

	/*********** initializing wights *******************/

	/***************** *****************/
	//cudaMemcpy(w, d_w, N * classes * sizeof(float), cudaMemcpyDeviceToHost);
	for(int k = 2; k < 3; k++) {
		for (int j = 0; j < height; ++j) {
			for (int i = 0; i < width; ++i) {
				cout <<  " " << w[ (j ) * height + (i ) + k * N];
			}
			cout << endl;
		}
	}
	cout << endl;
	cout << endl;
	cout << endl;

	/***************** *****************/

	/************************** *************************************************
	 *******************************
	 *******************************
	 *******************************  LOAD AND UPDATE
	 *******************************
	 *******************************
	 *  ********************* **************************************************/
	image.open(training_image_fn.c_str(), ios::in | ios::binary); // Binary image file
	label.open(training_label_fn.c_str(), ios::in | ios::binary ); // Binary label file

	// Reading file headers
	char number;
	for (int i = 1; i <= 16; ++i) {
		image.read(&number, sizeof(char));
	}
	for (int i = 1; i <= 8; ++i) {
		label.read(&number, sizeof(char));
	}

	for(int l = 0; l < 1000; l++) {

		/***************** Image Loading **********************/



		for (int sample = 1; sample <= nTraining; ++sample) {
			cout << "Sample ---------- **************" << sample << endl;

			input();
		}





		for (int i = 0; i < N; i++) {
			x[i] = (float)d[i % width][i / width];
		}

		//			cout << "Image:" << endl;
		//			for (int j = 0; j < height; ++j) {
		//				for (int i = 0; i < width; ++i) {
		//					cout << x[ (j ) * height + (i )];
		//				}
		//				cout << endl;
		//			}
		int hostNum[10];
		for(int j = 0; j < classes; j++)
			hostNum[j] = 0;
		hostNum[(int)inputNum] = 1;

		//	cout << "Label: ";
		//	for(int j = 0; j < classes; j++)
		//		cout << " " << hostNum[j];
		//	cout << endl;

		/***************** Image Loading **********************/


		/********* Multiplying ******************/

		hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);

		// Perform SAXPY on 1M elements
		int blockSize = 256;
		int numBlocks = (N + blockSize - 1) / blockSize;

		saxpy<<<numBlocks, blockSize>>>(N, 0.0f, d_x, d_w, d_sum);
		/********* Multiplying ******************/

		/***************** *****************/
//		cudaMemcpy(w, d_sum, N * classes * sizeof(float), cudaMemcpyDeviceToHost);
//		for(int k = 2; k < 3; k++) {
//			for (int j = 0; j < height; ++j) {
//				for (int i = 0; i < width; ++i) {
//					cout << " " << w[ (j ) * height + (i ) + k * N];
//				}
//				cout << endl;
//			}
//		}
//		cout << endl;
//		cout << endl;
//		cout << endl;
		/***************** *****************/


		/*********** Finding Softmax ************************/
		//cudaMemcpy(sum, d_sum, N*classes*sizeof(float), cudaMemcpyDeviceToHost);

		blockSize = 27 * 27;
		numBlocks = (classes + blockSize - 1) / blockSize;

		int max_index = 0;
		float total[10], summation = 0;
		for(int k = 0; k < classes; ++k) {
			h_index[0] = 0;
			hipMemcpy(d_index, h_index , classes * sizeof(float), hipMemcpyHostToDevice);

			sum_cuda<<<numBlocks, blockSize>>>(N, d_sum, d_index, k);
			hipMemcpy(h_index , d_index, classes * sizeof(int), hipMemcpyDeviceToHost);

			total[k] = h_index[0];
			//			cout << h_index[0] << endl;
			//			check( sum + k * N, N);
			//summation += total[k];
		}
		for(int k = 0; k < classes; ++k) {
			//total[k] = total[k] / summation;
			//				cout << total[k] << endl;

			max_index = total[k] > total[max_index] ? k : max_index;
		}
		for(int k = 0; k < classes; ++k) {
			//total[k] = total[k] / summation;
			//				cout << total[k] << endl;
			total[k] = total[k] - total[max_index];
			summation += total[k];
			//max_index = total[k] > total[max_index] ? k : max_index;
		}
		for(int k = 0; k < classes; ++k) {
			total[k] = total[k] / summation;
			//cout << total[k] << endl;
			//			total[k] = total[k] - total[max_index];
			//			summation += total[k];
			//max_index = total[k] > total[max_index] ? k : max_index;
		}

		/*********** Finding Softmax ************************/

		/***************** Checking the softmax **********/
		float temp = 0;
		for(int k = 0; k < classes; ++k) {
			temp += total[k];
		}
		cout << temp << " ---- " << max_index << endl;
		/***************** Checking the softmax **********/


		/*********** Finding Error ************************/
		//	cout << " Error : ";

		for(int k = 0; k < classes; k++) {
			err[k] = hostNum[k] - total[k];
			//					cout << " e: " << err[k];
		}
		//	cout << endl;

		hipMemcpy(d_err, err, classes * sizeof(float), hipMemcpyHostToDevice);
		/*********** Finding Error ************************/



		/************* Updating the weights *******************/
		blockSize = 27 * 27;
		numBlocks = (classes + blockSize - 1) / blockSize;
		updateWeights<<<numBlocks, blockSize>>>(N, d_err, d_w, d_x); // updateWeights(float n, float *err, float *w, float *x)


		/************* Updating the weights *******************/


	}

	report.close();
	image.close();
	label.close();

	/************************** *************************************************
	 *******************************
	 *******************************
	 *******************************  LOAD AND UPDATE
	 *******************************
	 *******************************
	 *  ********************* **************************************************/

	/***************** *****************/
	hipMemcpy(w, d_w, N * classes * sizeof(float), hipMemcpyDeviceToHost);
	for(int k = 2; k < 3; k++) {
		for (int j = 0; j < height; ++j) {
			for (int i = 0; i < width; ++i) {
				cout << " " << w[ (j ) * height + (i ) + k * N];
			}
			cout << endl;
		}
	}
	/***************** *****************/

	//	/***************** Checking the softmax **********/
	//	float temp = 0;
	//	for(int k = 0; k < classes; ++k) {
	//		temp += total[k];
	//	}
	//	cout << temp << "  " << max_index << endl;
	/***************** Checking the softmax **********/

}
